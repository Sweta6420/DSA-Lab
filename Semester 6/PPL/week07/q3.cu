#include "hip/hip_runtime.h"

#include <stdio.h>

__global__ void oneDconv(int *a,int *k,int *r, int as,int ks)
{	
	int gtid = blockIdx.x*blockDim.x+threadIdx.x;
	int h = ks/2;
	if (gtid<as) 
	{
		int result = 0, i, ii;
		for(i=0; i<ks; i++)
		{
			ii=gtid-h+i;
			if(ii>=0 && ii<=as) result=result+a[ii]*k[i];
		}
		r[gtid]=result;
	}
}
int main(void)
{
	int a[10]={1,2,3,4,5,6,7,8,9,10},k[3]={1,2,1},r[10];
	int as=10,ks=3,size1=as*sizeof(int),size2=ks*sizeof(int);
	int *da,*dk,*dr;
	hipMalloc((void **)&da,size1);
	hipMalloc((void **)&dk,size2);
	hipMalloc((void **)&dr,size1);
	hipMemcpy(da,a,size1,hipMemcpyHostToDevice);
	hipMemcpy(dk,k,size2,hipMemcpyHostToDevice);
	oneDconv<<<ceil(as/32.0),32>>>(da,dk,dr,as,ks);
	hipMemcpy(&r,dr,size1,hipMemcpyDeviceToHost);
	for(int i=0; i<as; i++) printf("%d ",r[i]);
	hipFree(da);
	hipFree(dk);
	hipFree(dr);
	return 0;
}