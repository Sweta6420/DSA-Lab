#include "hip/hip_runtime.h"

#include <stdio.h>

__global__ void vecadd(int *a,int *b,int *c, int n)
{	
	int gtid = blockIdx.x*blockDim.x+threadIdx.x;
	printf("a[%d] = %d, b[%d]= %d\n",gtid,a[gtid],gtid,b[gtid]);
	if (gtid<n) c[gtid] = a[gtid] + b[gtid];
}

int main(void)
{
	int a[100],b[100],c[100],n;
	printf("Enter n: ");
	scanf("%d",&n);
	printf("Enter A values: ");
	for(int i=0; i<n; i++) scanf("%d",&a[i]);
	printf("Enter B values: ");
	for(int i=0; i<n; i++) scanf("%d",&b[i]);
	int size = sizeof(int)*n;
	int *da,*db,*dc;
	hipMalloc((void **)&da,size);
	hipMalloc((void **)&db,size);
	hipMalloc((void **)&dc,size);
	hipMemcpy(da,a,size,hipMemcpyHostToDevice);
	hipMemcpy(db,b,size,hipMemcpyHostToDevice);
	vecadd<<<1,n>>>(da,db,dc,n);
	hipMemcpy(&c,dc,size,hipMemcpyDeviceToHost);
	for(int i=0; i<n; i++) printf("%d ",c[i]);
	hipFree(da);
	hipFree(db);
	hipFree(dc);
	return 0;
}