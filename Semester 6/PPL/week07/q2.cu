#include "hip/hip_runtime.h"

#include <stdio.h>

__global__ void vecadd(int *a,int *b,int *c, int n)
{	
	int gtid = blockIdx.x*blockDim.x+threadIdx.x;
	if (gtid<n) c[gtid] = a[gtid] + b[gtid];
}

int main(void)
{
	int a[500],b[500],c[500],n=500;
	for(int i=0; i<n; i++) a[i]=i+1;
	for(int i=0; i<n; i++) b[i]=i+1;
	int size = sizeof(int)*n;
	int *da,*db,*dc;
	hipMalloc((void **)&da,size);
	hipMalloc((void **)&db,size);
	hipMalloc((void **)&dc,size);
	hipMemcpy(da,a,size,hipMemcpyHostToDevice);
	hipMemcpy(db,b,size,hipMemcpyHostToDevice);
	vecadd<<<ceil(n/256.0),256>>>(da,db,dc,n);
	hipMemcpy(&c,dc,size,hipMemcpyDeviceToHost);
	for(int i=0; i<n; i++) printf("%d ",c[i]);
	hipFree(da);
	hipFree(db);
	hipFree(dc);
	return 0;
}