#include "hip/hip_runtime.h"

#include <stdio.h>
#include <math.h>

__global__ void sine(float *a,float *b,int n)
{	
	int gtid = blockIdx.x*blockDim.x+threadIdx.x;
	//printf("a[%d] = %.2f\n",gtid,a[gtid]);
	if (gtid<n) b[gtid] = sinf(a[gtid]);
}

int main(void)
{
	float a[10]={1.56,6.0,3.4,5.7,8.7,4.5,9.3,2.4,1.3,8.5},b[10];
	int n=10,size = sizeof(int)*n;
	float *da,*db;
	hipMalloc((void **)&da,size);
	hipMalloc((void **)&db,size);
	hipMemcpy(da,a,size,hipMemcpyHostToDevice);
	sine<<<ceil(n/256.0),256>>>(da,db,n);
	hipMemcpy(&b,db,size,hipMemcpyDeviceToHost);
	for(int i=0; i<n; i++) printf("%.2f ",b[i]);
	hipFree(da);
	hipFree(db);
	return 0;
}