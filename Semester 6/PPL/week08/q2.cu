#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#define MAX 1024
__global__ void modifyStr(char *A,char *B,int n)
{	
	int i = threadIdx.x;
	int start = n*(n+1)/2 - (i+1)*(i+2)/2;
	for(int j=0; j<=i; j++) 
	{	
		printf("%d %c %d\n",i,A[j],j+start);	
		B[j+start] = A[j];
	}
}

int main(void)
{
	char A[100],B[MAX],*d_A,*d_B;
	printf("Enter A: "); scanf("%s",A);
	int n = strlen(A);
	hipMalloc((void **)&d_A,strlen(A)*sizeof(char));
	hipMalloc((void **)&d_B,strlen(B)*sizeof(char));
	hipMemcpy(d_A,A,n*sizeof(char),hipMemcpyHostToDevice);
	modifyStr<<<1,n>>>(d_A,d_B,n);
	hipMemcpy(B,d_B,MAX*sizeof(char),hipMemcpyDeviceToHost);
	B[n*(n+1)/2]='\0';
	puts(B);
	hipFree(d_A);
	hipFree(d_B);
	return 0;
}