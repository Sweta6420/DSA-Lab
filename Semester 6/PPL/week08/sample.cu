#include "hip/hip_runtime.h"

#include <stdio.h>
#define N 1024
__global__ void countA(char *A,int *d_count)
{	
	int gtid = threadIdx.x;
	if(A[gtid]=='a') atomicAdd(d_count,1);
}

int main(void)
{
	char A[N], *d_A;
	int count=0, *d_count;
	printf("Enter a string: "); scanf("%s",A);
	hipEvent_t start,stop;
	hipEventCreate(&start); hipEventCreate(&stop);
	hipEventRecord(start,0);
	hipMalloc((void **)&d_A,strlen(A)*sizeof(char));
	hipMalloc((void **)&d_count,sizeof(int));
	hipMemcpy(d_A,A,strlen(A)*sizeof(char),hipMemcpyHostToDevice);
	hipMemcpy(d_count,&count,sizeof(int), hipMemcpyHostToDevice);
	countA<<<1,strlen(A)>>>(d_A,d_count);
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime,start,stop);
	hipMemcpy(&count,d_count,sizeof(int),hipMemcpyDeviceToHost);
	printf("A occurences: %d\n",count);
	printf("Time taken: %f",elapsedTime);
	hipFree(d_A);
	hipFree(d_count);
	return 0;
}