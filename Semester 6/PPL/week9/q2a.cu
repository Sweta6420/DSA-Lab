#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
__global__ void matmul(int *a, int *b, int *t,int q, int n)
{
	int r = threadIdx.x;
	for(int c=0; c<q; c++)
	{
		int sum=0;
		for(int k=0; k<n; k++) sum+=a[r*n+k]*b[k*q+c];
		t[r*q+c]=sum;
	}
}
int main(void)
{
	int *a, *b, *t, m,n,p,q;
	int *d_a,*d_b,*d_t;
	printf("m value: "); scanf("%d",&m);
	printf("n value: "); scanf("%d",&n);
	printf("p value: "); scanf("%d",&p);
	printf("q value: "); scanf("%d",&q);
	int size = sizeof(int)*m*n, size1 = sizeof(int)*p*q, size2=sizeof(int)*m*q;
	a = (int *) malloc(size);
	b = (int *) malloc(size1);
	t = (int *) malloc(size2);
	printf("Enter matrix A: ");
	for(int i=0; i<m*n; i++) scanf("%d",&a[i]);
	printf("Enter matrix B: ");
	for(int i=0; i<p*q; i++) scanf("%d",&b[i]);
	hipMalloc((void **) &d_a,size);
	hipMalloc((void **) &d_b,size1);
	hipMalloc((void **) &d_t,size2);
	hipMemcpy(d_a,a,size,hipMemcpyHostToDevice);
	hipMemcpy(d_b,b,size1,hipMemcpyHostToDevice);
	matmul<<<1,m>>>(d_a,d_b,d_t,q,n);
	hipMemcpy(t,d_t,size2,hipMemcpyDeviceToHost);
	printf("Result vector:\n");
	for(int i=0; i<m; i++)
	{
		for(int j=0; j<q; j++) printf("%d ",t[i*q+j]);
		printf("\n");
	}
	hipFree(d_a);
	hipFree(d_t);
	return 0;	
}