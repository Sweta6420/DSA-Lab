#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
__global__ void matadd(int *a, int *b, int *t,int m)
{
	int n=threadIdx.x,size=blockDim.x;
	for(int i=0; i<m; i++)
	{
		t[i*size+n] = a[i*size+n]+b[i*size+n];
		printf("Thread %d - %d\n",n,t[i*size+n]);
	}
}
int main(void)
{
	int *a, *b,*t, m,n;
	int *d_a,*d_b,*d_t;
	printf("m value: "); scanf("%d",&m);
	printf("n value: "); scanf("%d",&n);
	int size = sizeof(int)*m*n;
	a = (int *) malloc(size);
	b = (int *) malloc(size);
	t = (int *) malloc(size);
	printf("Enter matrix A: ");
	for(int i=0; i<m*n; i++) scanf("%d",&a[i]);
	printf("Enter matrix B: ");
	for(int i=0; i<m*n; i++) scanf("%d",&b[i]);
	hipMalloc((void **) &d_a,size);
	hipMalloc((void **) &d_b,size);
	hipMalloc((void **) &d_t,size);
	hipMemcpy(d_a,a,size,hipMemcpyHostToDevice);
	hipMemcpy(d_b,b,size,hipMemcpyHostToDevice);
	matadd<<<1,n>>>(d_a,d_b,d_t,m);
	hipMemcpy(t,d_t,size,hipMemcpyDeviceToHost);
	printf("Result vector:\n");
	for(int i=0; i<m; i++)
	{
		for(int j=0; j<n; j++) printf("%d ",t[i*n+j]);
		printf("\n");
	}
	hipFree(d_a);
	hipFree(d_t);
	return 0;	
}