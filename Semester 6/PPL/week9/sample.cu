#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
__global__ void transpose(int *a, int *t)
{
	int n = threadIdx.x, m=blockIdx.x, size=blockDim.x, size1=gridDim.x;
	t[n*size1+m] = a[m*size+n];
}
int main(void)
{
	int *a, *t, m,n;
	int *d_a,*d_t;
	printf("m value: "); scanf("%d",&m);
	printf("n value: "); scanf("%d",&n);
	int size = sizeof(int)*m*n;
	a = (int *) malloc(size);
	t = (int *) malloc(size);
	printf("Enter matrix: ");
	for(int i=0; i<m*n; i++) scanf("%d",&a[i]);
	hipMalloc((void **) &d_a,size);
	hipMalloc((void **) &d_t,size);
	hipMemcpy(d_a,a,size,hipMemcpyHostToDevice);
	transpose<<<m,n>>>(d_a,d_t);
	hipMemcpy(t,d_t,size,hipMemcpyDeviceToHost);
	printf("Result vector:\n");
	for(int i=0; i<n; i++)
	{
		for(int j=0; j<m; j++) printf("%d ",t[i*m+j]);
		printf("\n");
	}
	hipFree(d_a);
	hipFree(d_t);
	return 0;	
}