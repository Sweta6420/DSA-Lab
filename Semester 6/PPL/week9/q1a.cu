#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
__global__ void matadd(int *a, int *b, int *t,int n)
{
	int m=threadIdx.x;
	for(int i=0; i<n; i++)
	{
		t[m*n+i] = a[m*n+i]+b[m*n+i];
	}
}
int main(void)
{
	int *a, *b,*t, m,n;
	int *d_a,*d_b,*d_t;
	printf("m value: "); scanf("%d",&m);
	printf("n value: "); scanf("%d",&n);
	int size = sizeof(int)*m*n;
	a = (int *) malloc(size);
	b = (int *) malloc(size);
	t = (int *) malloc(size);
	printf("Enter matrix A: ");
	for(int i=0; i<m*n; i++) scanf("%d",&a[i]);
	printf("Enter matrix B: ");
	for(int i=0; i<m*n; i++) scanf("%d",&b[i]);
	hipMalloc((void **) &d_a,size);
	hipMalloc((void **) &d_b,size);
	hipMalloc((void **) &d_t,size);
	hipMemcpy(d_a,a,size,hipMemcpyHostToDevice);
	hipMemcpy(d_b,b,size,hipMemcpyHostToDevice);
	matadd<<<1,m>>>(d_a,d_b,d_t,n);
	hipMemcpy(t,d_t,size,hipMemcpyDeviceToHost);
	printf("Result vector:\n");
	for(int i=0; i<m; i++)
	{
		for(int j=0; j<n; j++) printf("%d ",t[i*n+j]);
		printf("\n");
	}
	hipFree(d_a);
	hipFree(d_t);
	return 0;	
}